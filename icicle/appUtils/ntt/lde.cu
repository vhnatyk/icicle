#include "hip/hip_runtime.h"
#pragma once
#ifndef LDE
#define LDE
#include <hip/hip_runtime.h>
#include "ntt.cuh"
#include "lde.cuh"
#include "../vector_manipulation/ve_mod_mult.cuh"
#include "../../utils/tr.cuh"
#include <assert.h>

/**
 * Interpolate a batch of polynomials from their evaluations on the same subgroup.
 * Note: this function does not preform any bit-reverse permutations on its inputs or outputs.
 * @param d_out The variable to write coefficients of the resulting polynomials into (the coefficients are in bit-reversed order if the evaluations weren't bit-reversed and vice-versa).
 * @param d_evaluations Input array of evaluations of all polynomials of type E (elements).
 * @param d_domain Domain on which the polynomials are evaluated. Must be a subgroup.
 * @param n Length of `d_domain` array, also equal to the number of evaluations of each polynomial.
 * @param batch_size The size of the batch; the length of `d_evaluations` is `n` * `batch_size`.
 */
template <typename E, typename S>
int interpolate_batch(E *d_out, E *d_evaluations, S *d_domain, unsigned n, unsigned batch_size)
{
  uint32_t logn = uint32_t(log(n) / log(2));
  hipMemcpy(d_out, d_evaluations, sizeof(E) * n * batch_size, hipMemcpyDeviceToDevice);

  int NUM_THREADS = min(n / 2, MAX_THREADS_BATCH);
  int chunks = max(int((n / 2) / NUM_THREADS), 1);
  int total_tasks = batch_size * chunks;
  int NUM_BLOCKS = total_tasks;
  int max_sharedmem = 512 * sizeof(E);
  int shared_mem = 2 * NUM_THREADS * sizeof(E); // TODO: calculator, as shared mem size may be more efficient less then max to allow more concurrent blocks on SM
  uint32_t logn_shmem = uint32_t(log(2 * NUM_THREADS) / log(2));
  // ntt_template_kernel_shared<<<NUM_BLOCKS, NUM_THREADS, shared_mem, 0>>>(d_out, 1 << logn_shmem, d_domain, n, total_tasks, 0, logn_shmem, false);

  // for (uint32_t s = logn_shmem; s < logn; s++) // TODO: this loop also can be unrolled
  //for (uint32_t s = logn; s > 0; s--) // TODO: this loop also can be unrolled
  for (uint32_t s = 0; s < logn; s++) // TODO: this loop also can be unrolled
  {
    ntt_template_kernel_bc<<<NUM_BLOCKS, NUM_THREADS>>>(d_out, n, d_domain, n, total_tasks, s, false);
  }

  NUM_BLOCKS = (n * batch_size + NUM_THREADS - 1) / NUM_THREADS;
  template_normalize_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(d_out, n * batch_size, S::inv_log_size(logn));
  return 0;
}

/**
 * Interpolate a polynomial from its evaluations on a subgroup.
 * Note: this function does not preform any bit-reverse permutations on its inputs or outputs.
 * @param d_out The variable to write coefficients of the resulting polynomial into (the coefficients are in bit-reversed order if the evaluations weren't bit-reversed and vice-versa).
 * @param d_evaluations Input array of evaluations that have type E (elements).
 * @param d_domain Domain on which the polynomial is evaluated. Must be a subgroup.
 * @param n Length of `d_evaluations` and the size `d_domain` arrays (they should have equal length).
 */
template <typename E, typename S>
int interpolate(E *d_out, E *d_evaluations, S *d_domain, unsigned n)
{
  return interpolate_batch<E, S>(d_out, d_evaluations, d_domain, n, 1);
}

template <typename E>
__global__ void fill_array(E *arr, E val, uint32_t n)
{
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < n)
  {
    arr[tid] = val;
  }
}

template <typename E, typename S>
__global__ void bench_mul_kernel(E a, S b, E *r, size_t n, size_t samples)
{
  // S f1 = group_gen;
  // S f2 = f1 * group_gen_inverse;

  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < n)
  {
    // int scalar_id = tid % n_scalars;
    // element_vec[tid] = scalar_vec[scalar_id] * element_vec[tid];

    S t;

    for (int s2 = 0; s2 < samples; s2++)
    {
      t = t * b;
    }

    t = a * t;

    if (tid == 0)
    {
      *r = t;
    }
  }
}

template <typename E, typename S>
__global__ void bench_add_kernel(E a, S b, E *r, size_t n, size_t samples)
{
  // S f1 = group_gen;
  // S f2 = f1 * group_gen_inverse;

  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < n)
  {
    // int scalar_id = tid % n_scalars;
    // element_vec[tid] = scalar_vec[scalar_id] * element_vec[tid];

    S t;
    // for (int s1 = 0; s1 < samples; s1++)
    // {
    for (int s2 = 0; s2 < samples; s2++)
    {
      t = t + b;
    }
    // }

    t = a + t;

    if (tid == 0)
    {
      *r = t;
    }
  }
}

/**
 * Evaluate a batch of polynomials on the same coset.
 * @param d_out The evaluations of the polynomials on coset `u` * `d_domain`.
 * @param d_coefficients Input array of coefficients of all polynomials of type E (elements) to be evaluated in-place on a coset.
 * @param d_domain Domain on which the polynomials are evaluated (see `coset` flag). Must be a subgroup.
 * @param domain_size Length of `d_domain` array, on which the polynomial is computed.
 * @param n The number of coefficients, which might be different from `domain_size`.
 * @param batch_size The size of the batch; the length of `d_coefficients` is `n` * `batch_size`.
 * @param coset The flag that indicates whether to evaluate on a coset. If false, evaluate on a subgroup `d_domain`.
 * @param coset_powers If `coset` is true, a list of powers `[1, u, u^2, ..., u^{n-1}]` where `u` is the generator of the coset.
 */
template <typename E, typename S>
int evaluate_batch(E *d_out, E *d_coefficients, S *d_domain, unsigned domain_size, unsigned n, unsigned batch_size, bool coset, S *coset_powers)
{
  uint32_t logn = uint32_t(log(domain_size) / log(2));
  if (domain_size > n)
  {
    // allocate and initialize an array of stream handles to parallelize data copying across batches
    hipStream_t *memcpy_streams = (hipStream_t *)malloc(batch_size * sizeof(hipStream_t));
    for (int i = 0; i < batch_size; i++)
    {
      hipStreamCreate(&(memcpy_streams[i]));

      hipMemcpyAsync(&d_out[i * domain_size], &d_coefficients[i * n], n * sizeof(E), hipMemcpyDeviceToDevice, memcpy_streams[i]);
      int NUM_THREADS = MAX_THREADS_BATCH;
      int NUM_BLOCKS = (domain_size - n + NUM_THREADS - 1) / NUM_THREADS;
      fill_array<E><<<NUM_BLOCKS, NUM_THREADS, 0, memcpy_streams[i]>>>(&d_out[i * domain_size + n], E::zero(), domain_size - n);

      hipStreamSynchronize(memcpy_streams[i]);
      hipStreamDestroy(memcpy_streams[i]);
    }
  }
  else
    hipMemcpy(d_out, d_coefficients, sizeof(E) * domain_size * batch_size, hipMemcpyDeviceToDevice);

  if (coset)
    batch_vector_mult(coset_powers, d_out, domain_size, batch_size);

  int NUM_THREADS = min(domain_size / 2, MAX_THREADS_BATCH);
  int chunks = max(int((domain_size / 2) / NUM_THREADS), 1);
  int total_tasks = batch_size * chunks;
  int NUM_BLOCKS = total_tasks;
  int max_sharedmem = 512 * sizeof(E);
  int shared_mem = (2 * NUM_THREADS) * sizeof(E); // TODO: calculator, as shared mem size may be more efficient less then max to allow more concurrent blocks on SM
  uint32_t logn_shmem = uint32_t(log(2 * NUM_THREADS) / log(2));
  // for (uint32_t s = logn - 1; s >= logn_shmem; s--) // TODO: this loop also can be unrolled
  for (uint32_t s = logn; s > 0; s--) // TODO: this loop also can be unrolled
  // for (uint32_t s = 0; s < logn; s++) // TODO: this loop also can be unrolled
  {
    // ntt_template_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(d_out, domain_size, d_domain, domain_size, total_tasks, s - 1, true);
    //  ntt_template_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(d_out, domain_size, d_domain, domain_size, total_tasks, s, false);

    ntt_template_kernel_bc<<<NUM_BLOCKS, NUM_THREADS>>>(d_out, n, d_domain, n, total_tasks, s - 1, true);
  }

  // uint32_t log2_num_blocks = (log(NUM_BLOCKS) / log(2));
  // uint32_t n_div_log2_blocks = (((1 << logn_shmem) >> (log2_num_blocks + 1)) - 1);
  // uint32_t num_blocks2x = NUM_BLOCKS * 2; // TODO: ? uint32_t

  // ntt_template_kernel_shared_rev<<<NUM_BLOCKS, NUM_THREADS, shared_mem, 0>>>(d_out, 1 << logn_shmem, d_domain, n / 2, total_tasks, 0, logn_shmem - 1, n_div_log2_blocks, num_blocks2x, (1 << logn_shmem) - 1);
  // ntt_template_kernel_shared<<<NUM_BLOCKS, NUM_THREADS, shared_mem, 0>>>(d_out, 1 << logn_shmem, d_domain, n, total_tasks, 0, logn_shmem, false);

  return 0;
}

///
/**
 * Evaluate a batch of polynomials on the same coset.
 * @param d_inout Input array of type E (elements)
 * @param d_twf Twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
 * @param n The size of single input.
 * @param batch_size The size of the batch; the length of `d_inout` is `n` * `batch_size`.
 */
template <typename E, typename S>
int ntt_batch_template(E *d_inout, S *d_twf, unsigned n, unsigned batch_size)
{
  uint32_t logn = uint32_t(log(n) / log(2));

  int NUM_THREADS = min(n / 2, MAX_THREADS_BATCH);
  int chunks = max(int((n / 2) / NUM_THREADS), 1);
  int total_tasks = batch_size * chunks;
  int NUM_BLOCKS = total_tasks;
  int max_sharedmem = 512 * sizeof(E);
  int shared_mem = (2 * NUM_THREADS) * sizeof(E); // TODO: calculator, as shared mem size may be more efficient less then max to allow more concurrent blocks on SM
  uint32_t logn_shmem = uint32_t(log(2 * NUM_THREADS) / log(2));
  // for (uint32_t s = logn - 1; s >= logn_shmem; s--) // TODO: this loop also can be unrolled
  // for (uint32_t s = 0; s < logn; s++) // TODO: this loop also can be unrolled
  for (uint32_t s = logn; s > 0; s--) // TODO: this loop also can be unrolled
  {
    ntt_template_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(d_inout, n, d_twf, n, total_tasks, s - 1, true);
  }

  // uint32_t log2_num_blocks = (log(NUM_BLOCKS) / log(2));
  // uint32_t n_div_log2_blocks = (((1 << logn_shmem) >> (log2_num_blocks + 1)) - 1);
  // uint32_t num_blocks2x = NUM_BLOCKS * 2; // TODO: ? uint32_t

  // ntt_template_kernel_shared_rev<<<NUM_BLOCKS, NUM_THREADS, shared_mem, 0>>>(d_inout, 1 << logn_shmem, d_twf, n / 2, total_tasks, 0, logn_shmem - 1, n_div_log2_blocks, num_blocks2x, (1 << logn_shmem) - 1);

  return 0;
}

template <typename E, typename S>
int ntt_batch_bc_template(E *d_inout, S *d_twf, unsigned n, unsigned batch_size, bool r, bool t)
{
  uint32_t logn = uint32_t(log(n) / log(2));

  int NUM_THREADS = min(n / 2, MAX_THREADS_BATCH);
  int chunks = max(int((n / 2) / NUM_THREADS), 1);
  int total_tasks = batch_size * chunks;
  int NUM_BLOCKS = total_tasks;
  int max_sharedmem = 512 * sizeof(E);
  int shared_mem = (2 * NUM_THREADS) * sizeof(E); // TODO: calculator, as shared mem size may be more efficient less then max to allow more concurrent blocks on SM
  uint32_t logn_shmem = uint32_t(log(2 * NUM_THREADS) / log(2));
  if (r)
  {
    // for (uint32_t s = logn - 1; s >= logn_shmem; s--) // TODO: this loop also can be unrolled
    // for (uint32_t s = 0; s < logn; s++) // TODO: this loop also can be unrolled
    for (uint32_t s = logn; s > 0; s--) // TODO: this loop also can be unrolled
    {
      ntt_template_kernel_bc<<<NUM_BLOCKS, NUM_THREADS>>>(d_inout, n, d_twf, n, total_tasks, s - 1, t);
    }
  }
  else
  {
    // for (uint32_t s = 0; s < logn; s++) // TODO: this loop also can be unrolled
    for (uint32_t s = 0; s < logn; s++) // TODO: this loop also can be unrolled
    {
      ntt_template_kernel_bc<<<NUM_BLOCKS, NUM_THREADS>>>(d_inout, n, d_twf, n, total_tasks, s, t);
    }
  }

  // uint32_t log2_num_blocks = (log(NUM_BLOCKS) / log(2));
  // uint32_t n_div_log2_blocks = (((1 << logn_shmem) >> (log2_num_blocks + 1)) - 1);
  // uint32_t num_blocks2x = NUM_BLOCKS * 2; // TODO: ? uint32_t

  // ntt_template_kernel_shared_rev<<<NUM_BLOCKS, NUM_THREADS, shared_mem, 0>>>(d_inout, 1 << logn_shmem, d_twf, n / 2, total_tasks, 0, logn_shmem - 1, n_div_log2_blocks, num_blocks2x, (1 << logn_shmem) - 1);

  return 0;
}

template <typename S>
int ntt_batch_bc(S *d_inout, S *d_twf, unsigned n, unsigned batch_size, bool r, bool t)
{
  return ntt_batch_bc_template(d_inout, d_twf, n, batch_size, r, t);
}

template <typename S>
int ntt_batch(S *d_inout, S *d_twf, unsigned n, unsigned batch_size)
{
  return ntt_batch_template(d_inout, d_twf, n, batch_size);
}

template <typename S>
int bailey_ntt(S *d_inout, S *d_twf, S *d_full_twf, unsigned n, unsigned batch_size, bool r1, bool t1, bool tt1, bool r2, bool t2, bool tt2)
{
  uint32_t logn = uint32_t(log(n) / log(2));

  dim3 threads(TILE_DIM, BLOCK_ROWS);
  dim3 blocks(batch_size / TILE_DIM, n / TILE_DIM);

  ntt_batch_bc(d_inout, d_twf, n, batch_size, r1, t1);

  transpose<<<blocks, threads>>>(d_inout);

  batch_mul_tw_ij<<<batch_size, n>>>(d_inout, d_full_twf, n, batch_size, logn, tt1, tt2);
  ntt_batch_bc(d_inout, d_twf, n, batch_size, r2, t2);

  transpose<<<blocks, threads>>>(d_inout);

  return 0;
}
///

/**
 * Evaluate a polynomial on a coset.
 * Note: this function does not preform any bit-reverse permutations on its inputs or outputs, so the order of outputs is bit-reversed.
 * @param d_out The evaluations of the polynomial on coset `u` * `d_domain`.
 * @param d_coefficients Input array of coefficients of a polynomial of type E (elements).
 * @param d_domain Domain on which the polynomial is evaluated (see `coset` flag). Must be a subgroup.
 * @param domain_size Length of `d_domain` array, on which the polynomial is computed.
 * @param n The number of coefficients, which might be different from `domain_size`.
 * @param coset The flag that indicates whether to evaluate on a coset. If false, evaluate on a subgroup `d_domain`.
 * @param coset_powers If `coset` is true, a list of powers `[1, u, u^2, ..., u^{n-1}]` where `u` is the generator of the coset.
 */
template <typename E, typename S>
int evaluate(E *d_out, E *d_coefficients, S *d_domain, unsigned domain_size, unsigned n, bool coset, S *coset_powers)
{
  return evaluate_batch<E, S>(d_out, d_coefficients, d_domain, domain_size, n, 1, coset, coset_powers);
}

template <typename S>
int interpolate_scalars(S *d_out, S *d_evaluations, S *d_domain, unsigned n)
{
  return interpolate(d_out, d_evaluations, d_domain, n);
}

template <typename S>
int interpolate_scalars_batch(S *d_out, S *d_evaluations, S *d_domain, unsigned n, unsigned batch_size)
{
  return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size);
}

template <typename E, typename S>
int interpolate_points(E *d_out, E *d_evaluations, S *d_domain, unsigned n)
{
  return interpolate(d_out, d_evaluations, d_domain, n);
}

template <typename E, typename S>
int interpolate_points_batch(E *d_out, E *d_evaluations, S *d_domain, unsigned n, unsigned batch_size)
{
  return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size);
}

template <typename S>
int evaluate_scalars(S *d_out, S *d_coefficients, S *d_domain, unsigned domain_size, unsigned n)
{
  S *_null = nullptr;
  return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null);
}

template <typename S>
int evaluate_scalars_batch(S *d_out, S *d_coefficients, S *d_domain, unsigned domain_size, unsigned n, unsigned batch_size)
{
  S *_null = nullptr;
  return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null);
}

template <typename E, typename S>
int evaluate_points(E *d_out, E *d_coefficients, S *d_domain, unsigned domain_size, unsigned n)
{
  S *_null = nullptr;
  return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null);
}

template <typename E, typename S>
int evaluate_points_batch(E *d_out, E *d_coefficients, S *d_domain,
                          unsigned domain_size, unsigned n, unsigned batch_size)
{
  S *_null = nullptr;
  return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null);
}

template <typename S>
int evaluate_scalars_on_coset(S *d_out, S *d_coefficients, S *d_domain,
                              unsigned domain_size, unsigned n, S *coset_powers)
{
  return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers);
}

template <typename E, typename S>
int evaluate_scalars_on_coset_batch(S *d_out, S *d_coefficients, S *d_domain, unsigned domain_size,
                                    unsigned n, unsigned batch_size, S *coset_powers)
{
  return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers);
}

template <typename E, typename S>
int evaluate_points_on_coset(E *d_out, E *d_coefficients, S *d_domain,
                             unsigned domain_size, unsigned n, S *coset_powers)
{
  return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers);
}

template <typename E, typename S>
int evaluate_points_on_coset_batch(E *d_out, E *d_coefficients, S *d_domain, unsigned domain_size,
                                   unsigned n, unsigned batch_size, S *coset_powers)
{
  return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers);
}

#endif