#include "hip/hip_runtime.h"
#ifndef _BLS12_381_LDE
#define _BLS12_381_LDE
#include <hip/hip_runtime.h>
#include "../../appUtils/ntt/lde.cu"
#include "../../appUtils/ntt/ntt.cuh"
#include "../../appUtils/vector_manipulation/ve_mod_mult.cuh"
#include "curve_config.cuh"

extern "C" BLS12_381::scalar_t *build_domain_cuda_bls12_381(uint32_t domain_size, uint32_t logn, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        if (inverse)
        {
            return fill_twiddle_factors_array(domain_size, BLS12_381::scalar_t::omega_inv(logn), stream);
        }
        else
        {
            return fill_twiddle_factors_array(domain_size, BLS12_381::scalar_t::omega(logn), stream);
        }
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return nullptr;
    }
}

extern "C" int ntt_cuda_bls12_381(BLS12_381::scalar_t *arr, uint32_t n, bool inverse, Decimation decimation, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_template<BLS12_381::scalar_t, BLS12_381::scalar_t>(arr, n, inverse, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());

        return -1;
    }
}

extern "C" int ecntt_cuda_bls12_381(BLS12_381::projective_t *arr, uint32_t n, bool inverse, Decimation decimation, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_template<BLS12_381::projective_t, BLS12_381::scalar_t>(arr, n, inverse, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ntt_batch_cuda_bls12_381(BLS12_381::scalar_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_batch_template<BLS12_381::scalar_t, BLS12_381::scalar_t>(arr, arr_size, batch_size, inverse, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

// extern "C" int ecntt_batch_cuda_bls12_381(BLS12_381::projective_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0)
// {
//     try
//     {
//         return ntt_end2end_batch_template<BLS12_381::projective_t, BLS12_381::scalar_t>(arr, arr_size, batch_size, inverse); // TODO: pass device_id
//     }
//     catch (const std::runtime_error &ex)
//     {
//         printf("error %s", ex.what());
//         return -1;
//     }
// }

extern "C" int interpolate_scalars_cuda_bls12_381(BLS12_381::scalar_t *d_out, BLS12_381::scalar_t *d_evaluations, BLS12_381::scalar_t *d_domain, unsigned n, unsigned device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BLS12_381::scalar_t *_null = nullptr;
        return interpolate(d_out, d_evaluations, d_domain, n, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_batch_cuda_bls12_381(BLS12_381::scalar_t *d_out, BLS12_381::scalar_t *d_evaluations, BLS12_381::scalar_t *d_domain, unsigned n,
                                                        unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BLS12_381::scalar_t *_null = nullptr;
        hipStreamCreate(&stream);
        return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

// extern "C" int interpolate_points_cuda_bls12_381(BLS12_381::projective_t *d_out, BLS12_381::projective_t *d_evaluations, BLS12_381::scalar_t *d_domain, unsigned n, size_t device_id = 0)
// {
//     try
//     {
//         return interpolate(d_out, d_evaluations, d_domain, n);
//     }
//     catch (const std::runtime_error &ex)
//     {
//         printf("error %s", ex.what());
//         return -1;
//     }
// }

// extern "C" int interpolate_points_batch_cuda_bls12_381(BLS12_381::projective_t *d_out, BLS12_381::projective_t *d_evaluations, BLS12_381::scalar_t *d_domain,
//                                                        unsigned n, unsigned batch_size, size_t device_id = 0)
// {
//     try
//     {
//         return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size);
//     }
//     catch (const std::runtime_error &ex)
//     {
//         printf("error %s", ex.what());
//         return -1;
//     }
// }

extern "C" int evaluate_scalars_cuda_bls12_381(BLS12_381::scalar_t *d_out, BLS12_381::scalar_t *d_coefficients, BLS12_381::scalar_t *d_domain,
                                               unsigned domain_size, unsigned n, unsigned device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BLS12_381::scalar_t *_null = nullptr;
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_batch_cuda_bls12_381(BLS12_381::scalar_t *d_out, BLS12_381::scalar_t *d_coefficients, BLS12_381::scalar_t *d_domain, unsigned domain_size,
                                                     unsigned n, unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BLS12_381::scalar_t *_null = nullptr;
        hipStreamCreate(&stream);
        auto result_code = evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null, 0);
        hipStreamDestroy(stream);
        return result_code;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int bailey_ntt_cuda_bls12_381(BLS12_381::scalar_t *d_inout, BLS12_381::scalar_t *d_twf, BLS12_381::scalar_t *d_full_twf, uint32_t n, uint32_t batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        auto result_code = bailey_ntt(d_inout, d_twf, d_full_twf, n, batch_size, stream); // TODO: pass device_id
        hipStreamDestroy(stream);
        return result_code;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int bench_fr_add_cuda(size_t device_id, size_t samples, size_t blocks, size_t threads)
{
    // auto &gpu = select_gpu(device_id);
    BLS12_381::scalar_t f1 = BLS12_381::scalar_t::omega(8); // TODO: any value, random
    BLS12_381::scalar_t f2 = BLS12_381::scalar_t::omega(7);

    BLS12_381::scalar_t h_answer;
    BLS12_381::scalar_t *d_answer;
    hipMalloc(&d_answer, sizeof(BLS12_381::scalar_t));

    bench_add_kernel<<<blocks, threads>>>(f1, f2, d_answer, (size_t)(blocks * threads), samples);
    hipDeviceSynchronize();

    hipMemcpy(&h_answer, d_answer, sizeof(BLS12_381::scalar_t), hipMemcpyDeviceToHost);
    hipFree(d_answer);
    return 0;
}

extern "C" int bench_fr_sub_cuda(size_t device_id, size_t samples)
{
    // auto &gpu = select_gpu(device_id);
    // fr_t f1 = group_gen;
    // fr_t f2 = f1 * group_gen_inverse;

    // fr_t t;

    // for (int s = 0; s < samples; s++)
    // {
    //     t = f1 - f2;
    // }

    // fr_t f = t;

    return 0;
}

extern "C" int bench_fr_mul_cuda(size_t device_id, size_t samples, size_t blocks, size_t threads)
{
    // auto &gpu = select_gpu(device_id);
    BLS12_381::scalar_t f1 = BLS12_381::scalar_t::omega(8); // TODO: any value, random
    BLS12_381::scalar_t f2 = BLS12_381::scalar_t::omega(7);

    BLS12_381::scalar_t h_answer;
    BLS12_381::scalar_t *d_answer;
    hipMalloc(&d_answer, sizeof(BLS12_381::scalar_t));

    bench_mul_kernel<<<blocks, threads>>>(f1, f2, d_answer, (size_t)(blocks * threads), samples);
    hipDeviceSynchronize();

    hipMemcpy(&h_answer, d_answer, sizeof(BLS12_381::scalar_t), hipMemcpyDeviceToHost);
    hipFree(d_answer);
    return 0;
}



extern "C" int evaluate_points_cuda_bls12_381(BLS12_381::projective_t *d_out, BLS12_381::projective_t *d_coefficients, BLS12_381::scalar_t *d_domain,
                                              unsigned domain_size, unsigned n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BLS12_381::scalar_t *_null = nullptr;
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_batch_cuda_bls12_381(BLS12_381::projective_t *d_out, BLS12_381::projective_t *d_coefficients, BLS12_381::scalar_t *d_domain, unsigned domain_size,
                                                    unsigned n, unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BLS12_381::scalar_t *_null = nullptr;
        hipStreamCreate(&stream);
        auto result_code = evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null, stream);
        hipStreamDestroy(stream);
        return result_code;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_cuda_bls12_381(BLS12_381::scalar_t *d_out, BLS12_381::scalar_t *d_coefficients, BLS12_381::scalar_t *d_domain, unsigned domain_size,
                                                        unsigned n, BLS12_381::scalar_t *coset_powers, unsigned device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_batch_cuda_bls12_381(BLS12_381::scalar_t *d_out, BLS12_381::scalar_t *d_coefficients, BLS12_381::scalar_t *d_domain, unsigned domain_size,
                                                              unsigned n, unsigned batch_size, BLS12_381::scalar_t *coset_powers, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_cuda_bls12_381(BLS12_381::projective_t *d_out, BLS12_381::projective_t *d_coefficients, BLS12_381::scalar_t *d_domain, unsigned domain_size,
                                                       unsigned n, BLS12_381::scalar_t *coset_powers, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream); // TODO: don't create if default was passed, destroy what was created, same applies to all calls
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_batch_cuda_bls12_381(BLS12_381::projective_t *d_out, BLS12_381::projective_t *d_coefficients, BLS12_381::scalar_t *d_domain, unsigned domain_size,
                                                             unsigned n, unsigned batch_size, BLS12_381::scalar_t *coset_powers, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ntt_inplace_batch_cuda_bls12_381(BLS12_381::scalar_t *d_inout, BLS12_381::scalar_t *d_twiddles,
                                                unsigned n, unsigned batch_size, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        BLS12_381::scalar_t *_null = nullptr;
        ntt_inplace_batch_template(d_inout, d_twiddles, n, batch_size, inverse, false, _null, stream, true);
        return hipSuccess; // TODO: we should implement this https://leimao.github.io/blog/Proper-CUDA-Error-Checking/
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int fast_ntt_batch_cuda_bls12_381(BLS12_381::scalar_t *d_inout, BLS12_381::scalar_t *d_twf, uint32_t n, uint32_t batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    return ntt_inplace_batch_cuda_bls12_381(d_inout, d_twf, n, batch_size, false, device_id, stream);
}

extern "C" int reverse_order_scalars_cuda_bls12_381(BLS12_381::scalar_t *arr, int n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order(arr, n, logn, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_batch_cuda_bls12_381(BLS12_381::scalar_t *arr, int n, int batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order_batch(arr, n, logn, batch_size, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_cuda_bls12_381(BLS12_381::projective_t *arr, int n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order(arr, n, logn, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

// extern "C" int reverse_order_points_cuda_bls12_381(BLS12_381::projective_t *arr, int n, size_t device_id = 0)
// {
//     try
//     {
//         uint32_t logn = uint32_t(log(n) / log(2));
//         reverse_order(arr, n, logn);
//         return 0;
//     }
//     catch (const std::runtime_error &ex)
//     {
//         printf("error %s", ex.what());
//         return -1;
//     }
// }

// extern "C" int reverse_order_points_batch_cuda_bls12_381(BLS12_381::projective_t *arr, int n, int batch_size, size_t device_id = 0)
// {
//     try
//     {
//         uint32_t logn = uint32_t(log(n) / log(2));
//         reverse_order_batch(arr, n, logn, batch_size);
//         return 0;
//     }
//     catch (const std::runtime_error &ex)
//     {
//         printf("error %s", ex.what());
//         return -1;
//     }
// }
#endif